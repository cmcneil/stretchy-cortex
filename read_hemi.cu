#include "hip/hip_runtime.h"
extern "C" {
  #include <gifti_io.h>
}

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <list>

#include <GL/glew.h>
#include <glfw3.h>
#include <glm/glm.hpp>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>


// CUDA helper functions
// #include <hip/hip_runtime_api.h>
// #include <helper_cuda_gl.h>

#include "shader.h"
#include "controls.h"

// using namespace glm;
using namespace std;
using std::vector;
using std::list;
using glm::vec3;
// using thrust::device_vector;
// using thrust::host_vector;

template <typename T>
struct devarray {
    size_t nelements;
    T *data;
};

__device__
void updateVertexPositionVerlet(vec3 *pos, vec3 *prev, vec3 force) {
  vec3 temp = *pos;
  float damping = 0.0f;
  float tstep = 0.1f;
  *pos = temp + (temp - *prev) * (1.0f - damping) + force * tstep;
  *prev = temp;
}

__device__
vec3 getAccelerationOnVertex(vec3 *pos, int idx, devarray neighbor_list) {
  vec3 acceleration = vec3(0.0f, 0.0f, 0.0f);
  vec3 mypos = pos[idx];
  float k = 0.01;
  for (int i=0; i < neighbor_list.nelements; i++) {
    vec3 nbr_pos = pos[neighbor_list.data[i]];
    float dist = glm::distance(nbr_pos, mypos);
    vec3 dir_to_nbr = glm::normalize(nbr_pos - mypos);
    acceleration += dir_to_nbr*dist*k;
  }
  return acceleration;
}

__global__
void simple_distortion(vec3 *pos, int total_pts, float t, devarray* adj_list,
                       vec3 *prevs) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    float pi = acosf(-1);

    if (idx >= total_pts) {
      return;
    }

    float period = 10.0f; //s
    float w = sinf(2*pi*(t + __int2float_rd(idx) / __int2float_rd(total_pts))/period) * 0.01f;

    vec3 accel = getAccelerationOnVertex(pos, idx, adj_list[idx]);
    updateVertexPositionVerlet(pos+idx, prevs+idx, accel);
    // vec3 mypos = pos[idx];
    // vec3 stretch_dir = vec3(0.0, 0.0, 0.0);
    // devarray neighbor_list = adj_list[idx];
    // for (int i=0; i < neighbor_list.nelements; i++) {
    //   unsigned int nbr_idx = neighbor_list.data[i];
    //   vec3 pos_nbr = pos[nbr_idx];
    //   auto dir_to_nbr = glm::normalize(pos_nbr - mypos);
    //   stretch_dir += dir_to_nbr * -0.1f * w;
    // }

    // calculate simple sine wave pattern
    // float freq = 4.0f;
    // float period = 10.0f; //s
    // float w = sinf(2*pi*(t + __int2float_rd(idx) / __int2float_rd(total_pts))/period) * 0.01f;

    // write output vertex
    // vec3
    // pos[idx] = vec3(mypos[0] + w, mypos[1], mypos[2]) + stretch_dir;
}

// static void APIENTRY openglCallbackFunction(
//   GLenum source,
//   GLenum type,
//   GLuint id,
//   GLenum severity,
//   GLsizei length,
//   const GLchar* message,
//   const void* userParam
// ){
//   (void)source; (void)type; (void)id;
//   (void)severity; (void)length; (void)userParam;
//   fprintf(stderr, "%s\n", message);
//   if (severity==GL_DEBUG_SEVERITY_HIGH) {
//     fprintf(stderr, "Aborting...\n");
//     abort();
//   }
// }
void freeDevArr(devarray* arr, size_t total_pts) {
  for (int i=0; i < total_pts; i++) {
    hipFree(arr[i].data);
  }
  // hipFree(arr);
}

devarray<unsigned int>* polysToAdjacencyList(vector<vec3> pts,
                                             vector<unsigned int> polys,
                                             devarray<unsigned int>* cuda_adj_list,
                                             devarray<float>* cuda_init_dist) {
  // Alright, this function is a bit grungy.
  // First, we make a nice, C++ typed data structure that makes sense for the
  // adjacency matrix (sparsely represented), and load the data into that:
  vector<list<unsigned int>> adjacency_list(pts.size());
  int n_edges = 0;
  for (int i=0; i < polys.size(); i += 3) {
    for (int j=0; j < 3; j++) {
      adjacency_list[polys[i + j]].push_back(polys[i + ((j+1) % 3)]);
      adjacency_list[polys[i + j]].push_back(polys[i + ((j+2) % 3)]);
    }
  }
  // We make sure to eliminate duplicate neighbors:
  for (int i=0; i < adjacency_list.size(); i++) {
    adjacency_list[i].sort();
    adjacency_list[i].unique();
    n_edges += adjacency_list[i].size();
  }

  // Now comes the grungy part. We can't really copy these C++ pointer-y data structures
  // directly to the GPU. So we use a struct that will allow us to keep track of
  // the length of a list (devarray), essentially replicating the functionality
  // of vector<>. We copy our data into a list of those, and then
  // we allocate and copy to GPU copies.
  // Because it's really a list of pointers, each pointer in the list has to be
  // allocated by CUDA and copied to the GPU.
  devarray<unsigned int>* temp_adjacency_list = new devarray<unsigned int>[adjacency_list.size()];
  devarray<float>* temp_dist_list = new devarray<float>[adjacency_list.size()];
  // For each row, copy the data, then get a GPU pointer to it.
  for (int i=0;i < adjacency_list.size(); i++) {
    auto temp_adj_data = new unsigned int[adjacency_list[i].size()];
    auto temp_dist_data = new float[adjacency_list[i].size()];
    int j = 0;
    for (auto e : adjacency_list[i]) {
      // Get the edge
      temp_adj_data[j] = e;
      // Get the initial edge length
      temp_dist_data[j] = glm::distance(pts[e], pts[i]);
      j++;
    }
    devarray<unsigned int> adj_row;
    size_t srow = sizeof(unsigned int) * adjacency_list[i].size();
    hipMalloc((void **) &adj_row.data, srow);
    hipMemcpy(adj_row.data, temp_adj_data, srow, hipMemcpyHostToDevice);
    adj_row.nelements = adjacency_list[i].size();
    temp_adjacency_list[i] = adj_row;

    devarray<float> dist_row;
    srow = sizeof(float) * adjacency_list[i].size();
    hipMalloc((void **) &dist_row.data, srow);
    hipMemcpy(dist_row.data, temp_dist_data, srow, hipMemcpyHostToDevice);
    dist_row.nelements = adjacency_list[i].size();
    temp_dist_list[i] = dist_row;
  }
  // Now, take the temp array (of structs containing GPU pointers)
  // we've built up, and copy it to the GPU.
  devarray* cuda_adjacency_list;
  hipMalloc((void **) &cuda_adjacency_list, sizeof(devarray)*adjacency_list.size());
  hipMemcpy(cuda_adjacency_list, temp_adjacency_list,
             sizeof(devarray)*adjacency_list.size(), hipMemcpyHostToDevice);

  return cuda_adjacency_list;
}

void printAdjList(vector<list<unsigned int>> l) {
  for (auto sublist : l) {
    cout << "row: ";
    for (auto idx : sublist) {
      cout << " " << idx;
    }
    cout << endl;
  }
}


class GLManager {
 public:
   GLManager(int _winx, int _winy) : winx(_winx), winy(_winy) {}
   int init();
   void run();
   void meshLoad(std::vector<glm::vec3>, std::vector<unsigned int>);
   void runCudaVertexUpdate();

 private:
   int winx, winy;
   GLFWwindow* window;

   int n_cortex_verts; // number of vertices in the mesh
   int n_poly_idx; // number of polys in the mesh

   // OpenGL objects:
   GLuint shaderProgramID;
   GLuint mesh_buffer; // Mesh Vertex VBO
   void *cuda_mesh_vbo_buffer = NULL;
   struct hipGraphicsResource *cuda_vbo_resource;
   GLuint poly_idx_buffer; // Polys (as index array) Index VBO
   GLuint mesh_vao; // Vertex Array for the above
   GLuint mvp_id;

   float g_fAnim = 0.0f;
   devarray* cuda_adjacency_list;

   vec3* cuda_prev_positions;
   InputHandler* input_handler;

};

int GLManager::init() {
  if( !glfwInit() )
  {
    fprintf(stderr, "Failed to initialize GLFW\n");
    return -1;
  }
  // glfwWindowHint(GLFW_SAMPLES, 4); // 4x antialiasing
  glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4); // We want OpenGL 3.3
  glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 5);
  // glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE); // To make MacOS happy; should not be needed
  glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
  glfwWindowHint(GLFW_OPENGL_DEBUG_CONTEXT, GL_TRUE);

  this->window = glfwCreateWindow(this->winx, this->winy, "Cortex Stretcher", NULL, NULL);
  if(this->window == NULL) {
    fprintf(stderr, "Failed to open GLFW window. If you have an Intel GPU, they are not 3.3 compatible. Try the 2.1 version of the tutorials.\n" );
    glfwTerminate();
    return -1;
  }
  glfwMakeContextCurrent(this->window);
  cout << "OpenGL version: " << glGetString(GL_VERSION);

  // Initialize CUDA
  cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());

  // Initialize GLEW
  glewExperimental=true;
  if (glewInit() != GLEW_OK) {
    fprintf(stderr, "Failed to initialize GLEW\n");
    return -1;
  }

  // Set GL debugging
  // GLint flags; glGetIntegerv(GL_CONTEXT_FLAGS, &flags);
  // if (flags & GL_CONTEXT_FLAG_DEBUG_BIT)
  // {
  //    cout << "Debug output enabled!" << endl;
  //    glEnable(GL_DEBUG_OUTPUT);
  //    glEnable(GL_DEBUG_OUTPUT_SYNCHRONOUS);
    //  glDebugMessageCallback(openglCallbackFunction, nullptr);
    //  glDebugMessageControl(GL_DONT_CARE, GL_DONT_CARE, GL_DONT_CARE, 0, nullptr, GL_TRUE);
  // }

  glfwSetInputMode(this->window, GLFW_STICKY_KEYS, GL_TRUE);
  glfwSetInputMode(this->window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
  glfwPollEvents();
  glfwSetCursorPos(this->window, 1024/2, 768/2);

  glClearColor(0.0f, 0.0f, 0.0f, 0.0f);

  this->shaderProgramID = LoadShaders("vertex.glsl", "fragment.glsl");
  // Get a handle for our "MVP" uniform
	this->mvp_id = glGetUniformLocation(this->shaderProgramID, "MVP");
  this->input_handler = new InputHandler(this->window, false);
}

void GLManager::run() {
  glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
  double lastTime = glfwGetTime();
  int nbFrames = 0;
  do {
      double currentTime = glfwGetTime();
      nbFrames++;
      if ( currentTime - lastTime >= 1.0 ){ // If last prinf() was more than 1 sec ago
         // printf and reset timer
         printf("%f ms/frame\n", 1000.0/double(nbFrames));
         nbFrames = 0;
         lastTime += 1.0;
      }
      // for (int i=0; i < 2000; i++) {
        this->runCudaVertexUpdate();
      // }
      hipDeviceSynchronize();
      // Swap buffers
      glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
      glUseProgram(this->shaderProgramID);

      // Compute the MVP matrix from keyboard and mouse input
  		this->input_handler->computeMatricesFromInputs();
  		glm::mat4 ProjectionMatrix = this->input_handler->getProjectionMatrix();
  		glm::mat4 ViewMatrix = this->input_handler->getViewMatrix();
  		glm::mat4 ModelMatrix = glm::mat4(1.0);
  		glm::mat4 MVP = ProjectionMatrix * ViewMatrix * ModelMatrix;
      glUniformMatrix4fv(this->mvp_id, 1, GL_FALSE, &MVP[0][0]);

      glBindVertexArray(this->mesh_vao);
      // glDrawArrays(GL_POINTS, 0, this->n_cortex_verts);
      glEnableClientState(GL_VERTEX_ARRAY);
      glDrawElements(GL_TRIANGLES, this->n_poly_idx, GL_UNSIGNED_INT, 0);
      glDisableClientState(GL_VERTEX_ARRAY);

      glfwSwapBuffers(this->window);
      glfwPollEvents();
      this->g_fAnim += 0.1f;
  // Check if the ESC key was pressed or the window was closed
  } while(glfwGetKey(window, GLFW_KEY_ESCAPE ) != GLFW_PRESS &&
          glfwWindowShouldClose(this->window) == 0);
  // Cleanup VBO and shader
	glDeleteBuffers(1, &this->mesh_buffer);
  glDeleteBuffers(1, &this->poly_idx_buffer);
	glDeleteProgram(this->shaderProgramID);
	glDeleteVertexArrays(1, &this->mesh_vao);

  // Free Allocated GPU Memory
  hipFree(this->cuda_mesh_vbo_buffer);
  // freeDevArr(this->cuda_adjacency_list, this->n_cortex_verts);
  hipFree(this->cuda_prev_positions);

  // hipProfilerStop();

	// Close OpenGL window and terminate GLFW
	glfwTerminate();
}

void GLManager::meshLoad(std::vector<glm::vec3> pts, std::vector<unsigned int> idx) {
  // CUDA STUFF
  // hipMalloc((void **)&d_vbo_buffer, pts.size() * sizeof(glm::vec3));
  this->cuda_adjacency_list = polysToAdjacencyList(pts, idx);
  hipMalloc((void **) &this->cuda_prev_positions, sizeof(vec3)*pts.size());
  hipMemcpy(this->cuda_prev_positions, &pts[0], sizeof(vec3)*pts.size(),
             hipMemcpyHostToDevice);

	glGenBuffers(1, &this->mesh_buffer);
	glBindBuffer(GL_ARRAY_BUFFER, this->mesh_buffer);
	glBufferData(GL_ARRAY_BUFFER, pts.size() * sizeof(glm::vec3),
               &pts[0], GL_DYNAMIC_DRAW);

 	glGenBuffers(1, &this->poly_idx_buffer);
 	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, this->poly_idx_buffer);
 	glBufferData(GL_ELEMENT_ARRAY_BUFFER, idx.size() * sizeof(unsigned int),
               &idx[0] , GL_STATIC_DRAW);

  glGenVertexArrays(1, &this->mesh_vao);
  glBindVertexArray(this->mesh_vao);
  glEnableVertexAttribArray(0);
  glBindBuffer(GL_ARRAY_BUFFER, this->mesh_buffer);
  // CUDA STUFF
  hipGraphicsGLRegisterBuffer(&this->cuda_vbo_resource, this->mesh_buffer,
                               cudaGraphicsMapFlagsWriteDiscard);
  glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, NULL);
  glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, this->poly_idx_buffer);

  this->n_poly_idx = idx.size();
  this->n_cortex_verts = pts.size();
}

void GLManager::runCudaVertexUpdate() {
  hipGraphicsMapResources(1, &this->cuda_vbo_resource, 0);
  size_t num_bytes;
  vec3 *dptr;
  hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                        this->cuda_vbo_resource);
  int num_pts = this-> n_cortex_verts;
  int num_sms = 28;
  int pts_per_block = num_pts / num_sms + 1;
  int num_blocks = 28;
  if (pts_per_block > 2048) {
    pts_per_block = 2048;
    num_blocks = num_pts / 2048 + 1;
  }
  int t = time(0);
  simple_distortion<<< num_blocks, pts_per_block >>>(dptr, num_pts, this->g_fAnim,
                                                 this->cuda_adjacency_list,
                                                 this->cuda_prev_positions);
}

std::vector<glm::vec3> giiToVertices(giiDataArray *d) {
  std::vector<glm::vec3> out_vertices;
  int c, size;
  float *newarr = new float[d->nvals];
  assert(d->datatype == NIFTI_TYPE_FLOAT32);
  gifti_copy_data_as_float(newarr, NIFTI_TYPE_FLOAT32, d->data, d->datatype, d->nvals);

  float maxx, maxy, minx, miny = 0;
  for (int i = 0; i < d->nvals; i += 3) {
    // printf("Vertex read: %.3f, %.3f\n", newarr[i], newarr[i+1]);
    if (newarr[i] > maxx) {
      maxx = newarr[i];
    } else if (newarr[i] < minx) {
      minx = newarr[i];
    }
    if (newarr[i+1] > maxy) {
      maxy = newarr[i+1];
    } else if (newarr[i+1] < miny) {
      miny = newarr[i+1];
    }
    out_vertices.push_back(glm::vec3(newarr[i], newarr[i+1], 0.0));
  }

  // Normalize the vector:
  for (int i = 0; i < out_vertices.size(); i++) {
    out_vertices[i] = glm::vec3(out_vertices[i].x / (maxx - minx),
                                out_vertices[i].y / (maxy - miny), 0.0);
  }
  return out_vertices;
}

std::vector<unsigned int> giiToIndices(giiDataArray *d) {
  std::vector<unsigned int> out_vertices;
  int c, size;
  assert(d->datatype == NIFTI_TYPE_INT32);
  float *newarr = new float[d->nvals];
  gifti_copy_data_as_float(newarr, NIFTI_TYPE_FLOAT32, d->data, d->datatype, d->nvals);
  cout << "reading indices" << endl;
  for (int i = 0; i < d->nvals; i++) { //d->nvals
    out_vertices.push_back(static_cast<unsigned int>(newarr[i]));
  }
  cout << "idx read" << endl;
  return out_vertices;
}

int main(int argc, char *argv[]) {
  gifti_image *out_im;
  cout << "input argument: " << argv[1] << endl;
  gifti_disp_lib_version();
  out_im = gifti_read_image(argv[1], 1);
  cout << "valid image: " << gifti_valid_gifti_image(out_im, 1) << endl;
  gifti_disp_LabelTable("Label Table:", &out_im->labeltable);
  for (int i = 0; i < out_im->numDA; i++) {
    gifti_disp_DataArray("datarray: ", out_im->darray[i], 1);
  }


  giiDataArray *pts = out_im->darray[0];
  giiDataArray *triangles = out_im->darray[1];
  gifti_disp_raw_data(triangles->data, triangles->datatype, 100, 1, stdout);

  cout << "successfully read data" << endl;

  GLManager* manager = new GLManager(1024, 768);
  manager->init();
  // glEnable(GL_DEBUG_OUTPUT);
  auto adj_list = polysToAdjacencyList(giiToVertices(pts), giiToIndices(triangles));
  // printAdjList(adj_list);
  manager->meshLoad(giiToVertices(pts), giiToIndices(triangles));

  cout << "loaded buffers " << endl;
  manager->run();

  return 0;
}
